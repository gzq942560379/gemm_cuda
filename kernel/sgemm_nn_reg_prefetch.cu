#include "hip/hip_runtime.h"

#include "util.h"
#include "kernel.h"
#include "matrix.h"
#include <hip/hip_runtime_api.h>
#include <>
#include <cstdio>

template <int Mtile, int Ntile, int Ktile>
__global__ void mat_mul_nn_kernel(const Matrix A, const Matrix B, Matrix C, const float alpha, const float beta)
{
	const int blockRow = blockIdx.x;
	const int blockCol = blockIdx.y;

	const int blockThreadNum = blockDim.x;
	const int blockThreadId = threadIdx.x;

	const int warpNum = blockThreadNum / warpSize;
	const int warpIdx = blockThreadId / warpSize;
	const int warpThreadId = blockThreadId % warpSize;

	const int warpRows = 2;
	const int warpCols = 4;
	const int warpRowId = warpIdx % warpRows;
	const int warpColId = warpIdx / warpRows;

	const int warpThreadRows = 8;
	const int warpThreadCols = 4;
	const int warpThreadRowId = warpThreadId % warpThreadRows;
	const int warpThreadColId = warpThreadId / warpThreadRows;

	const int warp_m = Mtile / warpRows;		  //  64 / 2 = 32
	const int Warp_n = Ntile / warpCols;		  //  64 / 4 = 16
	const int thread_m = warp_m / warpThreadRows; // 32 / 8 = 4
	const int thread_n = Warp_n / warpThreadCols; // 16 / 4 = 4

	Matrix Cblock = C.getSubMatrix(blockRow, blockCol, Mtile, Ntile);
	Matrix Cwarp = Cblock.getSubMatrix(warpRowId, warpColId, warp_m, Warp_n);
	Matrix Cthread = Cwarp.getSubMatrix(warpThreadRowId, warpThreadColId, thread_m, thread_n);

	const int ldc = thread_m;
	const int Csize = thread_m * thread_n;
	float Cvalue[Csize];

	memset(Cvalue, '\0', Csize * sizeof(float));

	const int lda = Mtile;
	const int ldb = Ktile;
	const int Asize = Mtile * Ktile;
	const int Bsize = Ktile * Ntile;

	__shared__ float As[2][Asize];
	__shared__ float Bs[2][Bsize];

	float Afragment[thread_m];
	float Bfragment[thread_n];

	int kb_pre = 0;
	int cur;
	int next = kb_pre & 1;
	Matrix Ablock_pre,Bblock_pre;
	if(kb_pre < (A.cols / Ktile)){
		Ablock_pre = A.getSubMatrix(blockRow, kb_pre, Mtile, Ktile);
		Bblock_pre = B.getSubMatrix(kb_pre, blockCol, Ktile, Ntile);
		for (int fid = blockThreadId; fid < Asize; fid += blockThreadNum)
			As[next][fid] = Ablock_pre.getValue(fid % lda, fid / lda);
		for (int fid = blockThreadId; fid < Bsize; fid += blockThreadNum)
			Bs[next][fid] = Bblock_pre.getValue(fid % ldb, fid / ldb);
	}

	for (int kb = 0; kb < (A.cols / Ktile); ++kb)
	{
		__syncthreads();
		const Matrix Ablock = Ablock_pre;
		const Matrix Bblock = Bblock_pre;
		kb_pre = kb + 1;
		cur = kb & 1;
		next = kb_pre & 1;
		if(kb_pre < (A.cols / Ktile)){
			Ablock_pre = A.getSubMatrix(blockRow, kb_pre, Mtile, Ktile);
			Bblock_pre = B.getSubMatrix(kb_pre, blockCol, Ktile, Ntile);
			for (int fid = blockThreadId; fid < Asize; fid += blockThreadNum)
				As[next][fid] = Ablock_pre.getValue(fid % lda, fid / lda);
			for (int fid = blockThreadId; fid < Bsize; fid += blockThreadNum)
				Bs[next][fid] = Bblock_pre.getValue(fid % ldb, fid / ldb);
		}
		const Matrix Awarp = Ablock.getSubMatrix(warpRowId, 0, warp_m, Ktile);
		const Matrix Bwarp = Bblock.getSubMatrix(0, warpColId, Ktile, Warp_n);
		const Matrix Athread = Awarp.getSubMatrix(warpThreadRowId, 0, thread_m, Ktile);
		const Matrix Bthread = Bwarp.getSubMatrix(0, warpThreadColId, Ktile, thread_n);
		for (int k = 0; k < Ktile; ++k)
		{
			for (int tr = 0; tr < thread_m; tr++)
				Afragment[tr] = As[cur][index2(k, tr, lda)];
			for (int tc = 0; tc < thread_n; tc++)
				Bfragment[tc] = Bs[cur][index2(tc, k, ldb)];
			for (int tc = 0; tc < thread_n; tc++)
				for (int tr = 0; tr < thread_m; tr++)
					Cvalue[index2(tc, tr, ldc)] += Afragment[tr] * Bfragment[tc];
		}
	}
	__syncthreads();
	for (int tc = 0; tc < thread_n; tc++)
	{
		for (int tr = 0; tr < thread_m; tr++)
		{
			Cthread.setValue(tr, tc, Cvalue[index2(tc, tr, ldc)] * alpha + Cthread.getValue(tr, tc) * beta);
		}
	}
}

blas_status sgemm_nn(int m, int n, int k, const float *alpha, const float *A, int lda, const float *B, int ldb, const float *beta, float *C, int ldc)
{
	const Matrix a(m, k, lda, const_cast<float*>(A));
	const Matrix b(k, n, ldb, const_cast<float*>(B));
	Matrix c(m, n, ldc, C);
	const int Mtile = 64;
	const int Ntile = 64;
	const int Ktile = 8;
	int thread_num = 256;
	mat_mul_nn_kernel<Mtile, Ntile, Ktile><<<dim3(m / Mtile, n / Ntile), dim3(thread_num)>>>(a, b, c, *alpha, *beta);
}